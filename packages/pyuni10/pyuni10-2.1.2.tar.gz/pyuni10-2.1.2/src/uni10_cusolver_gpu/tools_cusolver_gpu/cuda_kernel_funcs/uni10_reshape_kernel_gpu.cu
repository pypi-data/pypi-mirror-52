#include "hip/hip_runtime.h"
#include "uni10_sys_info/uni10_cusolver_gpu/uni10_device_const_cusolver_gpu.cuh"
#include "uni10_cusolver_gpu/tools_cusolver_gpu/cuda_kernel_funcs/uni10_kernel_gpu.h"

namespace uni10{

  namespace linalg_driver_internal{

    __global__ void _Reshape_kernel(double* oldElem, int bondNum, size_t elemNum, size_t* offset, double* newElem){

      //size_t oldIdx = blockIdx.y * MAXGRIDSIZE_X * MAXTHREADSPERBLOCK +  blockIdx.x * blockDim.x + threadIdx.x;
      size_t oldIdx = (blockIdx.y*(gridDim.x) +  blockIdx.x) * DCnst.block + threadIdx.x;
      size_t idx = oldIdx;
      size_t newIdx = 0;

      if(idx < elemNum){
        for(int i = 0; i < bondNum; i++){
          newIdx += (idx/offset[i]) * offset[bondNum + i];
          idx = idx % offset[i];
        }
        newElem[newIdx] = oldElem[oldIdx];
      }

    }

    void Reshape_kernel(double* oldElem, int bondNum, size_t elemNum, size_t* offset, double* newElem){

      size_t* D_offset;
      checkCudaErrors(hipMalloc((void**)&D_offset, 2 * sizeof(size_t) * bondNum));
      checkCudaErrors(hipMemcpy(D_offset, offset, 2 * sizeof(size_t) * bondNum, hipMemcpyHostToDevice));

      const MemoryConst &host_const = env_variables.GetSysInfo().host_const;
      uni10_uint64 NBlk = (elemNum + host_const.block - 1) / host_const.block;
      dim3 grid(NBlk%MAXGRIDSIZE_X_H, (NBlk + MAXGRIDSIZE_X_H - 1)/MAXGRIDSIZE_X_H);

      _Reshape_kernel<<<grid, host_const.block>>>(oldElem, bondNum, elemNum, D_offset, newElem);

      hipFree(D_offset);


    }

    __global__ void _Reshape_kernel(std::complex<double>* oldElem, int bondNum, size_t elemNum, size_t* offset, std::complex<double>* newElem){

      size_t oldIdx = (blockIdx.y*(gridDim.x) +  blockIdx.x) * DCnst.block + threadIdx.x;
      size_t idx = oldIdx;
      size_t newIdx = 0;

      if(idx < elemNum){
        for(int i = 0; i < bondNum; i++){
          newIdx += (idx/offset[i]) * offset[bondNum + i];
          idx = idx % offset[i];
        }
        newElem[newIdx] = oldElem[oldIdx];
      }

    }

    void Reshape_kernel(std::complex<double>* oldElem, int bondNum, size_t elemNum, size_t* offset, std::complex<double>* newElem){

      size_t* D_offset;
      checkCudaErrors(hipMalloc((void**)&D_offset, 2 * sizeof(size_t) * bondNum));
      checkCudaErrors(hipMemcpy(D_offset, offset, 2 * sizeof(size_t) * bondNum, hipMemcpyHostToDevice));


      const MemoryConst &host_const = env_variables.GetSysInfo().host_const;
      uni10_uint64 NBlk = (elemNum + host_const.block - 1) / host_const.block;
      dim3 grid(NBlk%MAXGRIDSIZE_X_H, (NBlk + MAXGRIDSIZE_X_H - 1)/MAXGRIDSIZE_X_H);

      _Reshape_kernel<<<grid, host_const.block>>>(oldElem, bondNum, elemNum, D_offset, newElem);


      hipFree(D_offset);

    }

  }

}
