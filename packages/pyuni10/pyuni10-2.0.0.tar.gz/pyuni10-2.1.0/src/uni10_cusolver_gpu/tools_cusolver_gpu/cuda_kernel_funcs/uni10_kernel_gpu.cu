#include "hip/hip_runtime.h"
#include "uni10_sys_info/uni10_cusolver_gpu/uni10_device_const_cusolver_gpu.cuh"
#include "uni10_cusolver_gpu/tools_cusolver_gpu/cuda_kernel_funcs/uni10_kernel_gpu.h"

namespace uni10{

  namespace linalg_driver_internal{

    // uni10_double64
    __global__ void _VectorMul_kernel(uni10_double64* Y, uni10_double64* X, uni10_uint64 N){

      size_t idx = (blockIdx.y*(gridDim.x) +  blockIdx.x) * DCnst.block + threadIdx.x;

      if(idx < N)
        Y[idx] *= X[idx];

    }

    void VectorMul_kernel(uni10_double64* Y, uni10_double64* X, uni10_uint64 N){

      const MemoryConst &host_const = env_variables.GetSysInfo().host_const;
      uni10_uint64 NBlk = (N + host_const.block - 1) / host_const.block;
      dim3 grid(NBlk%MAXGRIDSIZE_X_H, (NBlk + MAXGRIDSIZE_X_H - 1)/MAXGRIDSIZE_X_H);

      _VectorMul_kernel<<< grid, host_const.block >>>(Y , X , N);


    }

    __global__ void _VectorSum_kernel(uni10_double64 a, uni10_double64* X, uni10_uint64 N){

    }

    void VectorSum_kernel(uni10_double64 a, uni10_double64* X, uni10_uint64 N){

    }

    __global__ void _SetDiag_kernel(uni10_double64* ori_elem, uni10_double64* diag_elem, uni10_uint64 M, uni10_uint64 N, uni10_uint64 diag_N){


    }

    void SetDiag_kernel(uni10_double64* ori_elem, uni10_double64* diag_elem, uni10_uint64 M, uni10_uint64 N, uni10_uint64 diag_N){

    }

    __global__ void _GetDiag_kernel(uni10_double64* ori_elem, uni10_double64* diag_elem, uni10_uint64 M, uni10_uint64 N, uni10_uint64 diag_N){

    }

    void GetDiag_kernel(uni10_double64* ori_elem, uni10_double64* diag_elem, uni10_uint64 M, uni10_uint64 N, uni10_uint64 diag_N){

    }

    __global__ void _GetUpTri_kernel(uni10_double64* ori_elem, uni10_double64* tri_elem, uni10_uint64 m, uni10_uint64 n){

    }

    void GetUpTri_kernel(uni10_double64* ori_elem, uni10_double64* tri_elem, uni10_uint64 m, uni10_uint64 n){

    }

    __global__ void _GetDnTri_kernel(uni10_double64* ori_elem, uni10_double64* tri_elem, uni10_uint64 m, uni10_uint64 n){

    }

    void GetDnTri_kernel(uni10_double64* ori_elem, uni10_double64* tri_elem, uni10_uint64 m, uni10_uint64 n){

    }

    // uni10_complex128
    __global__ void _VectorMul_kernel(uni10_complex128 a, uni10_complex128* X, uni10_uint64 N){

    }

    void VectorMul_kernel(uni10_complex128 a, uni10_complex128* X, uni10_uint64 N){

    }

    __global__ void _VectorSum_kernel(uni10_complex128 a, uni10_complex128* X, uni10_uint64 N){

    }

    void VectorSum_kernel(uni10_complex128 a, uni10_complex128* X, uni10_uint64 N){

    }

    __global__ void _SetDiag_kernel(uni10_complex128* ori_elem, uni10_complex128* diag_elem, uni10_uint64 M, uni10_uint64 N, uni10_uint64 diag_N){

    }

    void SetDiag_kernel(uni10_complex128* ori_elem, uni10_complex128* diag_elem, uni10_uint64 M, uni10_uint64 N, uni10_uint64 diag_N){

    }

    __global__ void _GetDiag_kernel(uni10_complex128* ori_elem, uni10_complex128* diag_elem, uni10_uint64 M, uni10_uint64 N, uni10_uint64 diag_N){

    }

    void GetDiag_kernel(uni10_complex128* ori_elem, uni10_complex128* diag_elem, uni10_uint64 M, uni10_uint64 N, uni10_uint64 diag_N){

    }

    __global__ void _GetUpTri_kernel(uni10_complex128* ori_elem, uni10_complex128* tri_elem, uni10_uint64 m, uni10_uint64 n){

    }

    void GetUpTri_kernel(uni10_complex128* ori_elem, uni10_complex128* tri_elem, uni10_uint64 m, uni10_uint64 n){

    }

    __global__ void _GetDnTri_kernel(uni10_complex128* ori_elem, uni10_complex128* tri_elem, uni10_uint64 m, uni10_uint64 n){

    }

    void GetDnTri_kernel(uni10_complex128* ori_elem, uni10_complex128* tri_elem, uni10_uint64 m, uni10_uint64 n){

    }

    // Auxiliary double64-complex128 || complex128-double64
    __global__ void _UniElemCast_kernel(uni10_complex128* new_elem, uni10_double64* raw_elem, uni10_uint64 elemNum){

    }

    void UniElemCast_kernel(uni10_complex128* new_elem, uni10_double64* raw_elem, uni10_uint64 elemNum){

    }

    __global__ void _UniElemCast_kernel(uni10_double64* new_elem, uni10_complex128* raw_elem, uni10_uint64 elemNum){

    }

    void UniElemCast_kernel(uni10_double64* new_elem, uni10_complex128* raw_elem, uni10_uint64 elemNum){

    }

  }

}
