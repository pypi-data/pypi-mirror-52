#include "hip/hip_runtime.h"
#include "uni10_sys_info/uni10_cusolver_gpu/uni10_device_const_cusolver_gpu.cuh"
#include "uni10_cusolver_gpu/tools_cusolver_gpu/cuda_kernel_funcs/uni10_kernel_gpu.h"

namespace uni10{

  namespace linalg_driver_internal{

    __global__ void _VectorExp_kernel(uni10_double64 a, uni10_double64* X, uni10_uint64 N){

      uni10_uint64 idx = (blockIdx.y*(gridDim.x) +  blockIdx.x) * DCnst.block + threadIdx.x;

      if(idx < N)
        X[idx] = exp( a * X[idx]);

    }

    void VectorExp_kernel(uni10_double64 a, uni10_double64* X, uni10_uint64 N){
      const MemoryConst &host_const = env_variables.GetSysInfo().host_const;
      uni10_uint64 NBlk = (N + host_const.block - 1) / host_const.block;
      dim3 grid(NBlk%MAXGRIDSIZE_X_H, (NBlk + MAXGRIDSIZE_X_H - 1)/MAXGRIDSIZE_X_H);

      //dim3 gridSize(blockNum % MAXGRIDSIZE_X_H, (blockNum + MAXGRIDSIZE_X_H - 1) / MAXGRIDSIZE_X_H);

      _VectorExp_kernel<<<grid, host_const.block>>>(a, X, N);

    }

    __global__ void _VectorExp_kernel(uni10_complex128 a, uni10_complex128* X, uni10_uint64 N){

      //uni10_uint64 idx = blockIdx.y * MAXGRIDSIZE_X * MAXTHREADSPERBLOCK + blockIdx.x * blockDim.x + threadIdx.x;

      //if(idx < N)
      //  X[idx] = exp( a * X[idx]);

    }

    void VectorExp_kernel(uni10_complex128 a, uni10_complex128* X, uni10_uint64 N){


      uni10_error_msg(true, "%s", "Developing");

      const MemoryConst &host_const = env_variables.GetSysInfo().host_const;
      uni10_uint64 NBlk = (N + host_const.block - 1) / host_const.block;
      dim3 grid(NBlk%MAXGRIDSIZE_X_H, (NBlk + MAXGRIDSIZE_X_H - 1)/MAXGRIDSIZE_X_H);

      //dim3 gridSize(blockNum % MAXGRIDSIZE_X_H, (blockNum + MAXGRIDSIZE_X_H - 1) / MAXGRIDSIZE_X_H);
      _VectorExp_kernel<<<grid, host_const.block>>>(a, X, N);


    }

  }

}
