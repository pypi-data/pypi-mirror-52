#include "uni10_sys_info/uni10_cusolver_gpu/uni10_memory_const_cusolver_gpu.h"
#include "uni10_sys_info/uni10_cusolver_gpu/uni10_device_const_cusolver_gpu.cuh"

#include "uni10_error.h"


namespace uni10{

    //gpu device instance
    __device__ __constant__ MemoryConst DCnst;

    void MemConstToGPU(const MemoryConst &HCnst){
        checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(DCnst),&HCnst,sizeof(MemoryConst)) );
    }
    void MemConstFromGPU(MemoryConst &HCnst){
        checkCudaErrors( hipMemcpyFromSymbol(&HCnst,HIP_SYMBOL(DCnst),sizeof(MemoryConst)) );
    }

}; // End of uni10 namespace

